#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <cmath>		/* log2() */
#include <cstdint> 		/* int64_t, uint64_t */
#include <cstdlib>		/* srand(), rand() */
#include <ctime>		/* time() */
#include <iostream> 		/* std::cout, std::endl */
#include <time.h>
#include <stdlib.h>
#include "utils.h" 	//INCLUDE HEADER FILE
#include "ntt.h"

using namespace std;
/**
 * Return vector with each element of the input at its bit-reversed position
 *
 * @param vec The vector to bit reverse
 * @param n   The length of the vector, must be a power of two
 * @return    The bit reversed vector
 */
__host__ __device__ uint64_t* bit_reverse(uint64_t* vec, uint64_t n) {

	uint64_t num_bits = log2(n);

	uint64_t* result;
	result = (uint64_t*)malloc(n * sizeof(uint64_t));

	uint64_t reverse_num;
	for (uint64_t i = 0; i < n; i++) {

		reverse_num = 0;
		for (uint64_t j = 0; j < num_bits; j++) {

			reverse_num = reverse_num << 1;
			if (i & (1 << j)) {
				reverse_num = reverse_num | 1;
			}
		}

		result[reverse_num] = vec[i];

	}

	return result;
}

/**
 * Compare two vectors element-wise and return whether they are equivalent
 *
 * @param vec1	The first vector to compare
 * @param vec2 	The second vector to compare
 * @param n 	The length of the vectors
 * @param debug	Whether to print debug information (will run entire vector)
 * @return 	Whether the two vectors are element-wise equivalent
 */
bool compVec(uint64_t* vec1, uint64_t* vec2, uint64_t n, bool debug) {

	bool comp = true;
	for (uint64_t i = 0; i < n; i++) {

		if (vec1[i] != vec2[i]) {
			comp = false;

			if (debug) {
				std::cout << "(vec1[" << i << "] : " << vec1[i] << ")";
				std::cout << "!= (vec2[" << i << "] : " << vec2[i] << ")";
				std::cout << std::endl;
			}
			else {
				break;
			}
		}
	}

	return comp;
}

/**
 * Perform the operation 'base^exp (mod m)' using the memory-efficient method
 *
 * @param base	The base of the expression
 * @param exp	The exponent of the expression
 * @param m	The modulus of the expression
 * @return 	The result of the expression
 */
__host__ __device__ uint64_t modExp(uint64_t base, uint64_t exp, uint64_t m) {

	uint64_t result = 1;

	while (exp > 0) {

		if (exp % 2) {

			result = modulo(result * base, m);

		}

		exp = exp >> 1;
		base = modulo(base * base, m);
	}

	return result;
}

/**
 * Perform the operation 'base (mod m)'
 *
 * @param base	The base of the expression
 * @param m	The modulus of the expression
 * @return 	The result of the expression
 */
__host__ __device__ uint64_t modulo(int64_t base, int64_t m) {
	int64_t result = base % m;

	return (result >= 0) ? result : result + m;
}

/**
 * Print an array of arbitrary length in a readable format
 *
 * @param vec	The array to be displayed
 * @param n	The length of the array
 */

 //void printVec(uint64_t* vec, uint64_t n) {
 //
 //	std::cout << "[";
 //
 //	for (uint64_t i = 0; i < n; i++) {
 //
 //		std::cout << vec[i] << ",";
 //
 //	}
 //
 //	std::cout << "]" << std::endl;
 //}

 /**
  * Generate an array of arbitrary length containing random positive integers
  *
  * @param n	The length of the array
  * @param max	The maximum value for an array element [Default: RAND_MAX]
  */
__host__ __device__ uint64_t* randVec(uint64_t n, uint64_t max) {

	uint64_t* vec;

	vec = (uint64_t*)malloc(n * sizeof(uint64_t));

	srand(time(0));
	for (uint64_t i = 0; i < n; i++) {

		vec[i] = rand() % (max + 1);

	}

	return vec;
}

//NTT.cpp Code

/**
 * Perform an in-place iterative breadth-first decimation-in-time Cooley-Tukey NTT on an input vector and return the result
 *
 * @param vec 	The input vector to be transformed
 * @param n	The size of the input vector
 * @param p	The prime to be used as the modulus of the transformation
 * @param r	The primitive root of the prime
 * @param rev	Whether to perform bit reversal on the input vector
 * @return 	The transformed vector
 */
__global__ void kernelNTT_DIT(uint64_t* result, uint64_t* vec, uint64_t n, uint64_t p, uint64_t r, uint64_t* mpow, uint64_t* akp, uint64_t logged, bool rev) {
	/*int j = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;*/

	uint64_t m, k_, a, factor1, factor2;
	//if (rev) {
	//	result = bit_reverse(vec, n);
	//}
	//else {
	//	for (uint64_t i = 0; i < n; i++) {
	//		result[i] = vec[i];
	//	}
	//}


	for (uint64_t i = 1; i <= logged; i++) {

		m = mpow[i - 1];

		k_ = (p - 1) / m;
		a = modExp(r, k_, p);

		for (uint64_t j = 0; j < n; j += m) {

			for (uint64_t k = 0; k < m / 2; k++) {

				factor1 = result[j + k];
				factor2 = modulo(akp[(i-1)*m/2+k] * result[j + k + m / 2], p);

				result[j + k] = modulo(factor1 + factor2, p);
				result[j + k + m / 2] = modulo(factor1 - factor2, p);


			}
		}

	}
	return;
}

void cudahelp(uint64_t* result, uint64_t* vec, uint64_t n, uint64_t p, uint64_t r, uint64_t* mpow, uint64_t* akp, int logged, bool rev)
{


	uint64_t* dev_a = nullptr;
	uint64_t* dev_c = nullptr;
	uint64_t* dev_mpow = nullptr;
	uint64_t* dev_akp = nullptr;

	hipMalloc((void**)&dev_mpow, logged * sizeof(uint64_t));
	hipMalloc((void**)&dev_c, n * sizeof(uint64_t));
	hipMalloc((void**)&dev_a, n * sizeof(uint64_t));
	hipMalloc((void**)&dev_akp, 12*2048 * sizeof(uint64_t));

	hipMemcpy(dev_mpow, mpow, logged * sizeof(uint64_t), hipMemcpyHostToDevice);
	hipMemcpy(dev_a, vec, n * sizeof(uint64_t), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, result, n * sizeof(uint64_t), hipMemcpyHostToDevice);
	hipMemcpy(dev_akp, akp, 12*2048 * sizeof(uint64_t), hipMemcpyHostToDevice);


	//dim3 block(32,32);
	kernelNTT_DIT << <1, 1 >> > (dev_c, dev_a, n, p, r, dev_mpow,dev_akp, logged, false);
	hipDeviceSynchronize();

	hipMemcpy(result, dev_c, n * sizeof(uint64_t), hipMemcpyDeviceToHost);

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_mpow);

}

//MAIN.cpp



int main(int argc, char* argv[]) {
	using std::chrono::high_resolution_clock;
	using std::chrono::duration_cast;
	using std::chrono::duration;
	using std::chrono::milliseconds;

	auto t1 = high_resolution_clock::now();

	const uint64_t n = 4096;
	uint64_t p = 68719403009;
	uint64_t r = 36048964756;

	const int logged = log2(n);

	uint64_t vec[n];
	uint64_t* result;

	for (int i = 0; i < n; i++) {
		vec[i] = i;
	}

	uint64_t mpow[12];
	uint64_t k_, a;
	uint64_t akp[12*2048];

	for (uint64_t i = 1; i <= 12; i++) {

		mpow[i - 1] = pow(2, i);
		k_ = (p - 1) / mpow[i - 1];
		a = modExp(r, k_, p);
		for (uint64_t k = 0; k < mpow[i - 1] / 2; k++) {
			akp[(i-1)* mpow[i - 1]/2 +k] = modExp(a, k, p);
		}
	}
	result = bit_reverse(vec, n);
	cudahelp(result, vec, n, p, r, mpow,akp, logged, false);

	auto t2 = high_resolution_clock::now();
	duration<double, std::milli> ms_double = t2 - t1;

	std::cout << "[";
	for (uint64_t i = 0; i < n; i++) {

		std::cout << result[i] << ",";

	}
	std::cout << "]" << std::endl;

	std::cout << ms_double.count() << "ms";
	return 0;
}
