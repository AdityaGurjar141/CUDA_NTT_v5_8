#include "hip/hip_runtime.h"
﻿#include <cmath>		/* log2() */
#include <cstdint> 		/* int64_t, uint64_t */
#include <cstdlib>		/* srand(), rand() */
#include <ctime>		/* time() */
#include <iostream> 		/* std::cout, std::endl */

#include "C:\Users\macad\source\repos\CUDA_NTT_v5_8\Header\ntt.h" 	//INCLUDE HEADER FILE
#include "C:\Users\macad\source\repos\CUDA_NTT_v5_8\Header\utils.h" 	//INCLUDE HEADER FILE

#include <C:\Users\macad\source\repos\CUDA_NTT_v5_8\Header\helper_functions.h>
#include <C:\Users\macad\source\repos\CUDA_NTT_v5_8\Header\hip/hip_runtime_api.h>
#include <C:\Users\macad\source\repos\CUDA_NTT_v5_8\Header\hip/hip_runtime.h>


/**
 * Return vector with each element of the input at its bit-reversed position
 *
 * @param vec The vector to bit reverse
 * @param n   The length of the vector, must be a power of two
 * @return    The bit reversed vector
 */
uint64_t* bit_reverse(uint64_t* vec, uint64_t n) {

	uint64_t num_bits = log2(n);

	uint64_t* result;
	result = (uint64_t*)malloc(n * sizeof(uint64_t));

	uint64_t reverse_num;
	for (uint64_t i = 0; i < n; i++) {

		reverse_num = 0;
		for (uint64_t j = 0; j < num_bits; j++) {

			reverse_num = reverse_num << 1;
			if (i & (1 << j)) {
				reverse_num = reverse_num | 1;
			}
		}

		result[reverse_num] = vec[i];

	}

	return result;
}

/**
 * Compare two vectors element-wise and return whether they are equivalent
 *
 * @param vec1	The first vector to compare
 * @param vec2 	The second vector to compare
 * @param n 	The length of the vectors
 * @param debug	Whether to print debug information (will run entire vector)
 * @return 	Whether the two vectors are element-wise equivalent
 */
bool compVec(uint64_t* vec1, uint64_t* vec2, uint64_t n, bool debug) {

	bool comp = true;
	for (uint64_t i = 0; i < n; i++) {

		if (vec1[i] != vec2[i]) {
			comp = false;

			if (debug) {
				std::cout << "(vec1[" << i << "] : " << vec1[i] << ")";
				std::cout << "!= (vec2[" << i << "] : " << vec2[i] << ")";
				std::cout << std::endl;
			}
			else {
				break;
			}
		}
	}

	return comp;
}

/**
 * Perform the operation 'base^exp (mod m)' using the memory-efficient method
 *
 * @param base	The base of the expression
 * @param exp	The exponent of the expression
 * @param m	The modulus of the expression
 * @return 	The result of the expression
 */
uint64_t modExp(uint64_t base, uint64_t exp, uint64_t m) {

	uint64_t result = 1;

	while (exp > 0) {

		if (exp % 2) {

			result = modulo(result * base, m);

		}

		exp = exp >> 1;
		base = modulo(base * base, m);
	}

	return result;
}

/**
 * Perform the operation 'base (mod m)'
 *
 * @param base	The base of the expression
 * @param m	The modulus of the expression
 * @return 	The result of the expression
 */
uint64_t modulo(int64_t base, int64_t m) {
	int64_t result = base % m;

	return (result >= 0) ? result : result + m;
}

/**
 * Print an array of arbitrary length in a readable format
 *
 * @param vec	The array to be displayed
 * @param n	The length of the array
 */
void printVec(uint64_t* vec, uint64_t n) {

	std::cout << "[";
	for (uint64_t i = 0; i < n; i++) {

		std::cout << vec[i] << ",";

	}
	std::cout << "]" << std::endl;
}

/**
 * Generate an array of arbitrary length containing random positive integers
 *
 * @param n	The length of the array
 * @param max	The maximum value for an array element [Default: RAND_MAX]
 */
uint64_t* randVec(uint64_t n, uint64_t max) {

	uint64_t* vec;
	vec = (uint64_t*)malloc(n * sizeof(uint64_t));

	srand(time(0));
	for (uint64_t i = 0; i < n; i++) {

		vec[i] = rand() % (max + 1);

	}

	return vec;
}








//NTT.CPP code

/**
 * Perform an in-place iterative breadth-first decimation-in-time Cooley-Tukey NTT on an input vector and return the result
 *
 * @param vec 	The input vector to be transformed
 * @param n	The size of the input vector
 * @param p	The prime to be used as the modulus of the transformation
 * @param r	The primitive root of the prime
 * @param rev	Whether to perform bit reversal on the input vector
 * @return 	The transformed vector
 */




uint64_t* inPlaceNTT_DIT(uint64_t* vec, uint64_t n, uint64_t p, uint64_t r, bool rev) {

	uint64_t* result;
	uint64_t m, k_, a, factor1, factor2;

	result = (uint64_t*)malloc(n * sizeof(uint64_t));

	if (rev) {
		result = bit_reverse(vec, n);
	}
	else {
		for (uint64_t i = 0; i < n; i++) {
			result[i] = vec[i];
		}
	}


	for (uint64_t i = 1; i <= log2(n); i++) {

		m = pow(2, i);

		k_ = (p - 1) / m;
		a = modExp(r, k_, p);

		for (uint64_t j = 0; j < n; j += m) {

			for (uint64_t k = 0; k < m / 2; k++) {

				factor1 = result[j + k];
				factor2 = modulo(modExp(a, k, p) * result[j + k + m / 2], p);

				result[j + k] = modulo(factor1 + factor2, p);
				result[j + k + m / 2] = modulo(factor1 - factor2, p);

			}
		}

	}

	return result;

}
using namespace std;

int main(int argc, char* argv[]) {
	clock_t t;
	t = clock();
	const uint64_t n = 4096;
	uint64_t p = 68719403009;
	uint64_t r = 36048964756;

	uint64_t vec[n];

	for (int i = 0; i < n; i++) {
		vec[i] = i;
	}

	uint64_t* outVec = inPlaceNTT_DIT(vec, n, p, r);

	printVec(outVec, n);
	t = clock() - t;
	printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	return 0;

}
